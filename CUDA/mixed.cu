#include "hip/hip_runtime.h"
// Matthew Holmes - holmem4
// Davis Putnam Solver

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/find.h>
#include <iostream>
#include <string>
#include <fstream>
#include <cstdlib>
#include <sstream>
#include <ctime>
#include <sys/time.h>

using namespace std;

vector<thrust::device_vector<int> > reduce(vector<thrust::device_vector<int> > s, int currentReduction)
{
    vector<thrust::device_vector<int> > newS;
    #pragma omp parallel for shared(newS)
    for(unsigned int i = 0; i < s.size(); i++)
    {
        thrust::device_vector<int>::iterator iter;
        iter = thrust::find(s[i].begin(), s[i].end(), currentReduction);
        if(iter != s[i].end())
        {
            continue;
        }
        iter = thrust::find(s[i].begin(), s[i].end(), -currentReduction);
        if(iter != s[i].end())
        {
           s[i].erase(iter);
           s[i].shrink_to_fit();
        }
        #pragma omp critical
        newS.push_back(s[i]);
    }
    return newS;
}

bool satisfiable(vector<thrust::device_vector<int> > &s, int currentReduction)
{
    if(s.size() == 0)
    {
        return true;
    }
    bool test = true;
    #pragma omp parallel for
    for(unsigned int i = 0; i < s.size(); i++)
    {
        if(s[i].size() == 0)
        {
            test = false;
        }
    }

    if(!test)
        return test;

//    cout<<"WAITING PREALLOC"<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    vector<thrust::device_vector<int> > s1 = reduce(s, currentReduction);
    vector<thrust::device_vector<int> > s2 = reduce(s, -currentReduction);
//    cout<<"ALLOC OCCURED"<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    for(unsigned int i = 0; i < s.size(); i++)
    {
        s[i].clear();
        s[i].shrink_to_fit();
    }
    return satisfiable(s1, currentReduction + 1) || satisfiable(s2, currentReduction + 1);
}

vector<thrust::device_vector<int> > parseInput(string inputFile)
{
    ifstream openFile(inputFile.c_str());
    if(!openFile.is_open())
    {
        cerr<<"ERROR OPENING FILE \""<<inputFile<<"\""<<endl<<"PANIC"<<endl;
        exit(-1);
    }
    vector<thrust::device_vector<int> > newS;
    string buffer;
    while(getline(openFile, buffer))
    {
        if(buffer.length() == 0)
            continue;
        istringstream is(buffer);
        int i = 0;
        thrust::host_vector<int> temp;
        while( is >> i)
        {
            temp.push_back(i);
        }
        thrust::device_vector<int> temp2 = temp;
        newS.push_back(temp2);
    }
    return newS;
}

int main(int argc, char* argv[])
{
    if(argc != 2)
    {
        cout<<"Incorrect Arguments."<<endl;
        cout<<"Usage:"<<endl<<"\t"<<argv[0]<<" input_file"<<endl;
        return 1;
    }
    vector<thrust::device_vector<int> > initialState = parseInput(argv[1]);
    cout<<"Beginning Davis Putnam Procedure."<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    struct timeval begin, end;
    gettimeofday(&begin, NULL);
    if(satisfiable(initialState, 1))
    {
        cout<<"Argument is invalid."<<endl;
    }
    else
    {
        cout<<"Argument is valid."<<endl;
    }
    gettimeofday(&end, NULL);
    cout<<"MS ELAPSED: "<<((end.tv_sec - begin.tv_sec) * 1000) + ((end.tv_usec - begin.tv_usec) / 1000)<<endl;
    return 1;
}
