#include "hip/hip_runtime.h"
// Matthew Holmes - holmem4
// Davis Putnam Solver

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/find.h>
#include <iostream>
#include <string>
#include <fstream>
#include <cstdlib>
#include <sstream>
#include <ctime>
#include <sys/time.h>
#include <cstring>

using namespace std;

vector<thrust::device_vector<int> > reduce(vector<thrust::device_vector<int> > &s, int currentReduction)
{
    vector<thrust::device_vector<int> > newS;
    for(unsigned int i = 0; i < s.size(); i++)
    {
        thrust::device_vector<int>::iterator iter;
        iter = thrust::find(s[i].begin(), s[i].end(), currentReduction);
        if(iter != s[i].end())
        {
            continue;
        }
        iter = thrust::find(s[i].begin(), s[i].end(), -currentReduction);
        if(iter != s[i].end())
        {
           s[i].erase(iter);
           s[i].shrink_to_fit();
        }
        newS.push_back(s[i]);
    }
    return newS;
}

bool satisfiable(vector<thrust::device_vector<int> > &s, int currentReduction)
{
    if(s.size() == 0)
    {
        return true;
    }
    for(unsigned int i = 0; i < s.size(); i++)
    {
        if(s[i].size() == 0)
        {
            return false;
        }
    }

//    cout<<"WAITING PREALLOC"<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    vector<thrust::device_vector<int> > s1 = reduce(s, currentReduction);
    vector<thrust::device_vector<int> > s2 = reduce(s, -currentReduction);
//    cout<<"ALLOC OCCURED"<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    for(unsigned int i = 0; i < s.size(); i++)
    {
        s[i].clear();
        s[i].shrink_to_fit();
    }
    return satisfiable(s1, currentReduction + 1) || satisfiable(s2, currentReduction + 1);
}

vector<thrust::device_vector<int> > parseInput(string inputFile)
{
    ifstream openFile(inputFile.c_str());
    if(!openFile.is_open())
    {
        cerr<<"ERROR OPENING FILE \""<<inputFile<<"\""<<endl<<"PANIC"<<endl;
        exit(-1);
    }
    vector<thrust::device_vector<int> > newS;
    string buffer;
    while(getline(openFile, buffer))
    {
        if(buffer.length() == 0)
            continue;
        istringstream is(buffer);
        int i = 0;
        thrust::host_vector<int> temp;
        while( is >> i)
        {
            temp.push_back(i);
        }
        thrust::device_vector<int> temp2 = temp;
        newS.push_back(temp2);
    }
    return newS;
}

int main(int argc, char* argv[])
{
    if(argc != 2)
    {
        cout<<"Incorrect Arguments."<<endl;
        cout<<"Usage:"<<endl<<"\t"<<argv[0]<<" input_file"<<endl;
        return 1;
    }
    vector<thrust::device_vector<int> > initialState = parseInput(argv[1]);
    cout<<"Beginning Davis Putnam Procedure."<<endl;
//    std::cin.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    struct timeval begin, end;
    gettimeofday(&begin, NULL);
    if(satisfiable(initialState, 1))
    {
        cout<<"Argument is invalid."<<endl;
    }
    else
    {
        cout<<"Argument is valid."<<endl;
    }
    gettimeofday(&end, NULL);
    int mselapsed = ((end.tv_sec - begin.tv_sec) * 1000) + ((end.tv_usec - begin.tv_usec) / 1000);
    cout<<"MS ELAPSED: "<<mselapsed<<endl;
    ofstream outputFile;
    outputFile.open(strcat(argv[1], "_data_cuda.csv"), ios::out | ios::app);
    outputFile<<mselapsed<<",";
    outputFile.close();
    return 1;
}
